#include <vector>

#include "caffe/layers/base_data_layer.hpp"

namespace caffe {

template <typename Dtype>
void BasePrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Batch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[1]->ReshapeLike(batch->label_);
    // Copy the labels.
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[1]->mutable_gpu_data());
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}


//multilabel licheng 2018-03-15
template <typename Dtype>
void BasePrefetchingMultilabelDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  MultilabelBatch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),top[0]->mutable_gpu_data());
  if (this->output_labels_) 
  {
    // Reshape to loaded labels.
    // Copy the labels.
    if (batch->label_.count()>0)
    {
      top[1]->ReshapeLike(batch->label_);
      caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[1]->mutable_gpu_data());
    }
    if (batch->pts_.count()>0)
    {
      top[2]->ReshapeLike(batch->label_);
      caffe_copy(batch->pts_.count(), batch->pts_.gpu_data(),
        top[2]->mutable_gpu_data());
    }
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}


INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);
INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingMultilabelDataLayer);

}  // namespace caffe
