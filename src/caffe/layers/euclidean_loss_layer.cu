#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/euclidean_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void EuclideanLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      diff_.mutable_gpu_data());

  if(use_hard_mining_)
  {
    caffe_sqr(count,diff_.cpu_data(),diff_sqr_.mutable_cpu_data());
    const Dtype* begin = diff_sqr_.cpu_data();
    const Dtype* end = diff_sqr_.cpu_data()+count;
    std::vector<Dtype> diff_sqr_list(begin,end);
    int hard_size  = round((1-hard_ratio_)*count);
    std::sort(diff_sqr_list.begin(), diff_sqr_list.end());
    CHECK_LT(hard_size,count)<<"hard_size_ must be less than count"<<std::endl;
    threshold_ = diff_sqr_list[hard_size];
  }

  Dtype dot;
  caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  Dtype loss = dot / bottom[0]->num() / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void EuclideanLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      int count = bottom[0]->count();
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] / bottom[i]->num();
      caffe_gpu_axpby(
          count,              // count
          alpha,                              // alpha
          diff_.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[i]->mutable_gpu_diff());  // b
       if(use_hard_mining_)
        {
          const Dtype *diff_sqr_data=diff_sqr_.cpu_data();
          for (int ii = 0; ii < count; ++ii)
          { 
            if (*(diff_sqr_data+ii)<threshold_)
            {
              *(bottom[i]->mutable_cpu_diff()+ii)=Dtype(0);
            }
          }
        }

    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(EuclideanLossLayer);

}  // namespace caffe
